#include <hip/hip_runtime.h>
#include <openssl/sha.h>
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>

#define N 8192 // dimension of the matrices
#define cudaCheck(e)                                                       \
  do {                                                                     \
    hipError_t error = (e);                                               \
    if (error != hipSuccess) {                                            \
      fprintf(stderr, "CUDA error at line %d (%s): %s\n", __LINE__,        \
              __func__, error, hipGetErrorString(error));                 \
      exit(1);                                                             \
    }                                                                      \
  } while (0)

const unsigned int RAND_SEED = 2306212; // My student ID is SY2306212
const size_t size = N * N * sizeof(double);
double A[N][N], B[N][N], C[N][N];

__global__ void cudaMatmul( //! line:cuda-kernel
    double *d_C, double *d_A, double *d_B) {
  int i = blockIdx.y * blockDim.y + threadIdx.y; //! line:cuda-blk-th-1
  int j = blockIdx.x * blockDim.x + threadIdx.x; //! line:cuda-blk-th-2

  if (i < N && j < N) {
    double sum = 0.0f;
    for (int k = 0; k < N; k++) {
      sum += d_A[i * N + k] * d_B[k * N + j];
    }
    d_C[i * N + j] = sum;
  }
}

int main() {
  struct timeval time;
  double *d_A, *d_B, *d_C;
  size_t th_per_blk;
  char *th_per_blk_s = getenv("THREADS_PER_BLOCK");
  th_per_blk = th_per_blk_s ? atoi(th_per_blk_s) : 16;
  th_per_blk = th_per_blk > 0 ? th_per_blk : 16;

  srand(RAND_SEED);
  for (int i = 0; i < N; i++)
    for (int j = 0; j < N; j++) {
      A[i][j] = (double)rand() / RAND_MAX;
      B[i][j] = (double)rand() / RAND_MAX;
      C[i][j] = 0.0;
    }

  puts("Performing matrix multiplication");
  gettimeofday(&time, NULL);
  double start = time.tv_sec + time.tv_usec / 1e6;
  cudaCheck(hipMalloc((void **)&d_A, size)); //! line:cuda-malloc-1
  cudaCheck(hipMalloc((void **)&d_B, size)); //! line:cuda-malloc-2
  cudaCheck(hipMalloc((void **)&d_C, size)); //! line:cuda-malloc-3

  cudaCheck(hipMemcpy(d_A, A, size,             //! line:cuda-memcpy-1
                       hipMemcpyHostToDevice)); //! line:cuda-memcpy-2
  cudaCheck(hipMemcpy(d_B, B, size,             //! line:cuda-memcpy-3
                       hipMemcpyHostToDevice)); //! line:cuda-memcpy-4

  dim3 threadsPerBlock(th_per_blk, th_per_blk);          //! line:cuda-tpb
  dim3 blocksPerGrid((N + th_per_blk - 1) / th_per_blk,  //! line:cuda-bpg-1
                     (N + th_per_blk - 1) / th_per_blk); //! line:cuda-bpg-2

  cudaMatmul<<<blocksPerGrid, threadsPerBlock>>>( //! line:cuda-matmul-1
      d_C,                                        //! line:cuda-matmul-2
      d_A,                                        //! line:cuda-matmul-3
      d_B);                                       //! line:cuda-matmul-4

  cudaCheck(hipGetLastError());      //! line:cuda-check-last-err
  cudaCheck(hipDeviceSynchronize()); //! line:cuda-sync

  cudaCheck(hipMemcpy(C, d_C, size,             //! line:cuda-memcpy-5
                       hipMemcpyDeviceToHost)); //! line:cuda-memcpy-6

  cudaCheck(hipFree(d_A)); //! line:cuda-free-1
  cudaCheck(hipFree(d_B)); //! line:cuda-free-2
  cudaCheck(hipFree(d_C)); //! line:cuda-free-3

  gettimeofday(&time, NULL);
  double end = time.tv_sec + time.tv_usec / 1e6;
  printf("Elapsed time: %.6f seconds, SHA1: ", end - start);

  unsigned char hash[SHA_DIGEST_LENGTH];
  SHA1((const unsigned char *)C, sizeof(C), hash);
  for (int i = 0; i < SHA_DIGEST_LENGTH; i++)
    printf("%02x%c", hash[i], i + 1 < SHA_DIGEST_LENGTH ? ' ' : '\n');
  return 0;
}
